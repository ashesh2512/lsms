#include "hip/hip_runtime.h"
/* -*- mode: C++; c-file-style: "bsd"; c-basic-offset: 2; indent-tabs-mode: nil -*- */

// test the inverion algorithm for multiple scattering codes for the solution of
// tau = (1 - tG)^-1 t
// where t is a block diagonal matrix
// note that the diagonal blocks G_ii == 0

#include <stdio.h>

#include "Complex.hpp"
#include "Matrix.hpp"
#include <vector>
#include <chrono>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h>

#include "inversionTest_cuda.hpp"

/*
class DeviceData
{
public:
  std::vector<hipDoubleComplex *> tMatrices;
  hipDoubleComplex *tMatrixStore;
  hipDoubleComplex *tau;
  hipDoubleComplex *tau00;
  hipDoubleComplex *m;
  hipDoubleComplex *G0;
  int *ipiv;
  int *info;
};
*/

void allocDeviceData(DeviceData &d, int blockSize, int numBlocks)
{
  int n=blockSize*numBlocks;
  hipMalloc((void**)&d.m, n*n*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.G0, n*n*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tau, n*blockSize*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tau00, blockSize*blockSize*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tMatrixStore, blockSize*blockSize*numBlocks*sizeof(hipDoubleComplex));
  d.tMatrices.resize(numBlocks);
  for(int i=0; i<numBlocks; i++)
    d.tMatrices[i] = &d.tMatrixStore[blockSize * blockSize * i];
  hipMalloc((void**)&d.ipiv, n*sizeof(int));
  hipMalloc((void**)&d.info, sizeof(int));
}

void freeDeviceData(DeviceData &d)
{
  hipFree(d.tau);
  hipFree(d.tau00);
  hipFree(d.m);
  hipFree(d.G0);
  hipFree(d.tMatrixStore);
  hipFree(d.ipiv);
  hipFree(d.info);
}

// #include "makegij_new.cpp"

void usage_cuda(const char *name)
{
  printf("usage: %s <matrix type> [options]\n",name);
  printf("  matrix type: 1: 1-tG and G, t Hilbert matrices, options: <block size> <num blocks>\n");
  printf("               2: 1-tG and G = -1, t = 1\n");
}

#define IDX(i, j, lDim) (((j)*(lDim))+i)

template <typename T>
void zeroMatrixCuda(T *devM, int lDim, int nCol)
{
//  for(int i=0; i<m.n_row(); i++)
//    for(int j=0; j<m.n_col(); j++)
//      m(i,j) = 0.0;
  hipMemset(devM, 0, lDim*nCol*sizeof(T));
}

template <typename T>
__global__ void setDiagonalKernel(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = val;
  }
}

template <typename T>
__global__ void addDiagonalKernel(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = hipCadd(devM[IDX(i, i, lDim)], val);
  }
}

template <typename T>
void unitMatrixCuda(T *devM, int lDim, int nCol)
{
  zeroMatrixCuda(devM, lDim, nCol);
  setDiagonalKernel<<<nCol,1>>>(devM, lDim, nCol, 1.0);
}

/*
Real matrixDistance(Matrix<Complex> &a, Matrix<Complex> &b)
{
  Real d;

  for(int i=0; i<a.n_col(); i++)
    for(int j=0; j<a.n_row(); j++)
      d += ((a(i,j)-b(i,j)) * std::conj(a(i,j)-b(i,j))).real();
  
  return std::sqrt(d);
}
*/

template <typename T>
__global__ void makeHilbertMatrixKernel(T *devM, int lDim, int nCol)
{
  int i = blockIdx.x;
  if(i<lDim)
  {
    for(int j=0; j<nCol; j++)
      devM[IDX(i,j, lDim)] = 1.0/(Complex(i+j+1));
  }
}

template <typename T>
__global__ void zeroDiagonalBlocksKernel(T *devM, int lDim, int nCol, int blockSize)
{
  int iBlock = blockIdx.x;
  int jBlock = blockIdx.y;
  if(iBlock<nCol/blockSize)
    if(jBlock<nCol/blockSize)
    {
      int ii=iBlock*blockSize;
      int jj=jBlock*blockSize;
      for(int i=0; i<std::min(blockSize, nCol-ii); i++)
        for(int j=0; j<std::min(blockSize, nCol-jj); j++)
          devM[IDX(ii+i, jj+j, lDim)] = 0.0;
    }
}

template <typename T>
void zeroDiagonalBlocksCuda(T *devM, int lDim, int nCol, int blockSize)
{
  zeroDiagonalBlocksKernel<<<nCol/blockSize,nCol/blockSize>>>(devM, lDim, nCol, blockSize);
}

void makeType1MatrixGPU(hipblasHandle_t cublasHandle, DeviceData &d, int blockSize, int numBlocks)
{
  int n=blockSize*numBlocks;
  hipDoubleComplex one = {1.0, 0.0};
  hipDoubleComplex mone = {-1.0, 0.0};
  hipDoubleComplex zero = {0.0, 0.0};
  std::vector<hipDoubleComplex *> ts(numBlocks*numBlocks);
  std::vector<hipDoubleComplex *> G0s(numBlocks*numBlocks);
  std::vector<hipDoubleComplex *> ms(numBlocks*numBlocks);

  for(int iBlock=0; iBlock<numBlocks; iBlock++)
  {
    for(int jBlock=0; jBlock<numBlocks; jBlock++)
    {
      ts[iBlock + jBlock*numBlocks] = d.tMatrices[iBlock];
      G0s[iBlock + jBlock*numBlocks] = &d.G0[IDX(iBlock*blockSize,jBlock*blockSize,n)];
      ms[iBlock + jBlock*numBlocks] = &d.m[IDX(iBlock*blockSize,jBlock*blockSize,n)];
    }
  }

  hipblasZgemmBatched(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, blockSize, blockSize, blockSize, &mone,
                                  &ts[0], blockSize,
                                  &G0s[0], n,
                                  &zero,
                                  &ms[0], n, 
                                  numBlocks*numBlocks);
  addDiagonalKernel<<<n,1>>>(d.m, n, n, one);
}

void transferMatrixToGPU(hipDoubleComplex *devM, Matrix<Complex> &m)
{
  hipMemcpy(devM, &m(0,0), m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
}

void transferMatrixFromGPU(Matrix<Complex> &m, hipDoubleComplex *devM)
{
  hipMemcpy(&m(0,0), devM,  m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}

__global__ void copyTMatrixToTau(hipDoubleComplex *tau, hipDoubleComplex *t, int blockSize, int numBlocks)
{
  int i = blockIdx.x;
  int n = blockSize*numBlocks;
  if(i < blockSize)
  {
    for(int j=0; j<blockSize; j++)
      tau[IDX(i,j,n)] = t[IDX(i,j,blockSize)];
  }
}

__global__ void copyTauToTau00(hipDoubleComplex *tau00, hipDoubleComplex *tau, int blockSize, int numBlocks)
{
  int i = blockIdx.x;
  int n = blockSize*numBlocks;
  if(i < blockSize)
  {
    for(int j=0; j<blockSize; j++)
      tau00[IDX(i,j,blockSize)] = tau[IDX(i,j,n)];
  }
}

void solveTau00zgetrf_cublas(hipblasHandle_t cublasHandle, DeviceData &d,
                             Matrix<Complex> &tau00, int blockSize, int numBlocks)
{
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *Aarray[1], *Barray[1];
 
  // printf("zero Matrix\n"); 
  zeroMatrixCuda(d.tau, blockSize*numBlocks, blockSize);
  // printf("copyTMatrixToTau\n");
  copyTMatrixToTau<<<blockSize,1>>>(d.tau, d.tMatrices[0], blockSize, numBlocks);

  Barray[0] = d.tau;

  Aarray[0] = d.m;
  
  int n = blockSize * numBlocks;
  int *ipivArray=d.ipiv;
  int *infoArray=d.info;
  int info;

  // printf("hipblasZgetrfBatched\n");
  hipblasZgetrfBatched(cublasHandle, n, Aarray, n, ipivArray, infoArray, 1);
  // printf("hipblasZgetrsBatched\n");

  hipblasZgetrsBatched(cublasHandle, HIPBLAS_OP_N, n, blockSize, Aarray, n, ipivArray,
                      Barray, n, &info, 1);

  // copy result into tau00
  // printf("copyTauToTau00\n");
  copyTauToTau00<<<blockSize,1>>>(d.tau00, d.tau, blockSize, numBlocks);
  // printf("transferMatrixFromGPU\n");
  transferMatrixFromGPU(tau00, d.tau00);
}

void initCuda(hipblasHandle_t &cublasHandle)
{
  hipblasCreate(&cublasHandle);
}

void finalizeCuda(hipblasHandle_t &cublasHandle)
{
  hipblasDestroy(cublasHandle);
}


