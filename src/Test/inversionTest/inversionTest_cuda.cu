#include "hip/hip_runtime.h"
/* -*- mode: C++; c-file-style: "bsd"; c-basic-offset: 2; indent-tabs-mode: nil -*- */

// test the inverion algorithm for multiple scattering codes for the solution of
// tau = (1 - tG)^-1 t
// where t is a block diagonal matrix
// note that the diagonal blocks G_ii == 0

#include <stdio.h>

#include "Complex.hpp"
#include "Matrix.hpp"
#include <vector>
#include <chrono>
#include <ctime>


class DeviceData
{
public:
  std::vector<hipDoubleComplex *> tMatrices;
  hipDoubleComplex *tMatrixStore;
  hipDoubleComplex *tau;
  hipDoubleComplex *tau00;
  hipDoubleComplex *m;
  hipDoubleComplex *G0;
  int *ipiv;
  int *info;
};

void allocDeviceData(DeviceData &d, int blockSize, int numBlocks)
{
  int n=blockSize*numBlocks;
  hipMalloc((void**)&d.m, n*n*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.G0, n*n*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tau, n*blockSize*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tau00, blockSize*blockSize*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tMatrixStore, blockSize*blockSize*numBlocks*sizeof(hipDoubleComplex));
  tMatrices.resize(numBlocks);
  for(int i=0; i<numBlocks; i++)
    tMatrices[i] = &tMatrixStore[blockSize * blockSize * i];
  hipMalloc((void**)&ipiv, n*sizeof(int));
  hipMalloc((void**)&info, sizeof(int));
}

void freeDeviceData(DeviceData &d)
{
  hipFree(d.tau);
  hipFree(d.tau00);
  hipFree(d.m);
  hipFree(d.G0);
  hipFree(d.tMatrixStore);
  hipFree(d.ipiv);
  hipFree(d.info);
}

// #include "makegij_new.cpp"

void usage(const char *name)
{
  printf("usage: %s <matrix type> [options]\n",name);
  printf("  matrix type: 1: 1-tG and G, t Hilbert matrices, options: <block size> <num blocks>\n");
  printf("               2: 1-tG and G = -1, t = 1\n");
}

#define IDX(i, j, lDim) (((j)*(lDim))+i)

template <typename T>
void zeroMatrixCuda(T *devM, int lDim, int nCol)
{
//  for(int i=0; i<m.n_row(); i++)
//    for(int j=0; j<m.n_col(); j++)
//      m(i,j) = 0.0;
  hipMemset(devM, 0, lDim*nCol*sizeof<T>);
}

template <typename T>
__global__ void setDiagonalKernel(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = val;
  }
}

template <typename T>
void unitMatrixCuda(T *devM, int lDim, int nCol)
{
  zeroMatrixCuda(T, lDim, nCol);
  setDiagonalKernel<<<nCol,1>>>(devM, lDim, nCol, 1.0);
}

Real matrixDistance(Matrix<Complex> &a, Matrix<Complex> &b)
{
  Real d;

  for(int i=0; i<a.n_col(); i++)
    for(int j=0; j<a.n_row(); j++)
      d += ((a(i,j)-b(i,j)) * std::conj(a(i,j)-b(i,j))).real();
  
  return std::sqrt(d);
}


template <typename T>
__global__ void makeHilbertMatrixKernel(T *devM, int lDim, int nCol)
{
  int i = blockIdx.x;
  if(i<lDim)
  {
    for(int j=0; j<nCol; j++)
      devM[IDX(i,j, lDim)] = 1.0/(Complex(i+j+1));
}

template <typename T>
__global__ void zeroDiagonalBlocksKernel(T *devM, int lDim, int nCol, int blockSize)
{
  int iBlock = blockIdx.x;
  int jBlock = blockIdx.y;
  if(iBlock<nCol/blockSize)
    if(jBlock<nCol/blockSize)
    {
      int ii=iBlock*blockSize;
      int jj=jBlock*blockSize;
      for(int i=0; i<std::min(blockSize, nCol-ii); i++)
        for(int j=0; j<std::min(blockSize, nCol-jj); j++)
          m[IDX(ii+i, jj+j, lDim)] = 0.0;
    }
}

template <typename T>
void zeroDiagonalBlocksCuda(T *devM, int lDim, int nCol, int blockSize)
{
  zeroDiagonalBlocksKernel<<<nCol/blockSize,nCol/blockSize>>>(devM, lDim, nCol, blockSize);
}

// type 1 matrix:
//
void makeType1Matrix(Matrix<Complex> &m, Matrix<Complex> &G0, std::vector<Matrix<Complex> > &tMatrices, int blockSize, int numBlocks)
{
  int n = m.n_row();
  Complex mone = -1.0;
  Complex zero = 0.0;
  // unitMatrix(m);
  // loop over the blocks to build -tG
  // m_ij <- t_i G_ij
  for(int iBlock=0; iBlock<numBlocks; iBlock++)
    for(int jBlock=0; jBlock<numBlocks; jBlock++)
    {
      // ZGEMM(TRANSA,TRANSB,M,N,K,ALPHA, A,LDA, B,LDB, BETA,C,LDC)
      // C := alpha*op( A )*op( B ) + beta*C,
      BLAS::zgemm_("N","N",&blockSize,&blockSize,&blockSize,&mone,
             &tMatrices[iBlock](0,0), &blockSize,
             &G0(iBlock*blockSize,jBlock*blockSize), &n,
             &zero, &m(iBlock*blockSize,jBlock*blockSize), &n);
    }
  // add unit matrix
  for(int i=0; i<m.n_row(); i++)
    m(i,i) = 1.0 + m(i,i);
}

void makeType1MatrixGPU(hipblasHandle_t cublasHandle, DeviceData &d, int blockSize, int numBlocks)
{
  int n=blockSize*numBlocks;
  hipDoubleComplex mone = -1.0;
  hipDoubleComplex zero = 0.0;
  std::vector<hipDoubleComplex *> ts(numBlocks*numBlocks);
  std::vector<hipDoubleComplex *> G0s(numBlocks*numBlocks);
  std::vector<hipDoubleComplex *> ms(numBlocks*numBlocks);

  for(int iBlock=0; iBlock<numBlocks; iBlock++)
    for(int jBlock=0; jBlock<numBlocks; jBlock++)
    {
      ts[iBlock + jBlock*numBlocks] = &d.tMatrices[iBlock];
      G0s[iBlock + jBlock*numBlocks] = &d.G0[IDX(iBlock*blockSize,jBlock*blockSize,n)];
      m[iBlock + jBlock*numBlocks] = &d.m[IDX(iBlock*blockSize,jBlock*blockSize,n)];
    }

  hipblasZgemmBatched(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, blockSize, blockSize, blockSize,
                                  &mone,
                                  const hipDoubleComplex *Aarray[], int lda,
                                  const hipDoubleComplex *Barray[], int ldb,
                                  const hipDoubleComplex *beta,
                                  hipDoubleComplex *Carray[], int ldc, 
                                  int batchCount)
}

void transferMatrixToGPU(Complex *devM, Matrix<Complex> &m)
{
  hipMemcpy(devM, &m(0,0), m.l_dim()*n_col()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
}

void transferMatrixFromGPU(Matrix<Complex> &m, Complex *devM)
{
  hipMemcpy(&m(0,0), devM,  m.l_dim()*n_col()*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}

__global__ void copyTMatrixToTau(hipDoubleComplex *tau, hipDoubleComplex *t, int blockSize, int numBlocks)
{
  int i = blockIdx.x;
  int n = blockSize*numBlocks;
  if(i < blockSize)
  {
    for(int j=0; j<blockSize; j++)
      tau[IDX(i,j,n)] = t[IDX(i,j,blockSize)];
  }
}

__global__ void copyTauToTau00(hipDoubleComplex *tau00, hipDoubleComplex *tau, int blockSize, int numBlocks)
{
  int i = blockIdx.x;
  int n = blockSize*numBlocks;
  if(i < blockSize)
  {
    for(int j=0; j<blockSize; j++)
      tau00[IDX(i,j,blockSize)] = tau[IDX(i,j,n)];
  }
}

void solveTau00zgetrf_cublas(hipblasHandle_t cublasHandle, DeviceData &d,
                             Matrix<Complex> &tau00, int blockSize, int numBlocks)
{
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *Aarray[1], *Barray[1];
  
  zeroMatrixGPU(d.tau);
  copyTMatrixToTau<<<blockSize,1>>>(d.tau, d.tMatrices[0], blockSize, numBlocks);

  Barray[0] = d.tau;
  
  int n = blockSize * numBlocks;
  int *ipivArray, *infoArray;
  hipMalloc((void**)&ipiv, n * sizeof(int));
  hipMalloc((void**)&infoArray, 1 * sizeof(int));

  hipblasZgetrfBatched(cublasHandle, n, Aarray, n, ipivArray, infoArray, 1);  
  hipblasZgetrsBatched(cublasHandle, HIPBLAS_OP_N, n, blockSize, Aarray, n, ipivArray,
                      Barray, n, infoArray, 1);

  // copy result into tau00
  copyTauToTau00<<<blockSize,1>>>(d.tau00, d.tau, blockSize, numBlocks);
  transferMatrixFromGPU(tau00, d.tau00);
}

void initCuda(hipblasHandle_t &cublasHandle)
{
  hipblasCreate(&cublasHandle);
}

void finalizeCuda(hipblasHandle_t &cublasHandle)
{
  hipblasDestroy(cublasHandle);
}


