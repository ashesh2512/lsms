#include "hip/hip_runtime.h"
/* -*- mode: C++; c-file-style: "bsd"; c-basic-offset: 2; indent-tabs-mode: nil -*- */

#include "linearSolvers.hpp"

#include <stdio.h>

#include "Complex.hpp"
#include "Matrix.hpp"
#include <vector>

#define IDX(i, j, lDim) (((j)*(lDim))+(i))

template <typename T>
void zeroMatrixCuda(T *devM, int lDim, int nCol)
{
//  for(int i=0; i<m.n_row(); i++)
//    for(int j=0; j<m.n_col(); j++)
//      m(i,j) = 0.0;
  hipMemset(devM, 0, lDim*nCol*sizeof(T));
}

template <typename T>
__global__ void setDiagonalKernelCuda(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = val;
  }
}

template <typename T>
__global__ void addDiagonalKernelCuda(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = hipCadd(devM[IDX(i, i, lDim)], val);
  }
}

template <typename T>
void unitMatrixCuda(T *devM, int lDim, int nCol)
{
  zeroMatrixCuda(devM, lDim, nCol);
  setDiagonalKernelCuda<<<nCol,1>>>(devM, lDim, nCol, 1.0);
}

template <typename T>
__global__ void zeroDiagonalBlocksKernelCuda(T *devM, int lDim, int nCol, int blockSize)
{
  int iBlock = blockIdx.x;
  int jBlock = blockIdx.y;
  if(iBlock<nCol/blockSize)
    if(jBlock<nCol/blockSize)
    {
      int ii=iBlock*blockSize;
      int jj=jBlock*blockSize;
      for(int i=0; i<std::min(blockSize, nCol-ii); i++)
        for(int j=0; j<std::min(blockSize, nCol-jj); j++)
          devM[IDX(ii+i, jj+j, lDim)] = 0.0;
    }
}


void transferMatrixToGPUCuda(hipDoubleComplex *devM, Matrix<Complex> &m)
{
  hipMemcpy(devM, &m(0,0), m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
}

void transferMatrixFromGPUCuda(Matrix<Complex> &m, hipDoubleComplex *devM)
{
  hipMemcpy(&m(0,0), devM,  m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}

__global__ void copyTMatrixToTauCuda(hipDoubleComplex *tau, hipDoubleComplex *t, int kkrsz, int nrmat)
{
  int i = blockIdx.x;
  if(i < kkrsz)
  {
    for(int j=0; j<kkrsz; j++)
      tau[IDX(i,j,nrmat)] = t[IDX(i,j,kkrsz)];
  }
}

__global__ void copyTauToTau00Cuda(hipDoubleComplex *tau00, hipDoubleComplex *tau, int kkrsz, int nrmat)
{
  int i = blockIdx.x;
  if(i < kkrsz)
  {
    for(int j=0; j<kkrsz; j++)
      tau00[IDX(i,j,kkrsz)] = tau[IDX(i,j,nrmat)];
  }
}

void solveTau00zgetrf_cublas(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                             Complex &tMatrix, Complex *devM,
                             Matrix<Complex> &tau00)
{
  hipblasHandle_t cublasHandle = DeviceStorage::getCublasHandle()
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *Aarray[1], *Barray[1];

  hipDoubleComplex *devTau = d.getDevTau();
  hipDoubleComplex *devTau00 = d.getDevTau00();
  // printf("zero Matrix\n");
  zeroMatrixCuda(devTau, nrmat_ns, kkrzs_ns);
  // printf("copyTMatrixToTau\n");
  copyTMatrixToTauCuda<<<blockSize,1>>>(devTau, tMatrix, kkrsz_ns, nrmat_ns);

  Barray[0] = devTau;

  Aarray[0] = devM;
  
  int *ipivArray=d.getDevIpiv;
  int *infoArray=d.info;
  int info;

  // printf("hipblasZgetrfBatched\n");
  hipblasZgetrfBatched(cublasHandle, nrmat_ns, Aarray, nrmat_ns, ipivArray, infoArray, 1);
  // printf("hipblasZgetrsBatched\n");

  hipblasZgetrsBatched(cublasHandle, HIPBLAS_OP_N, nrmat_ns, kkrzs_ns, Aarray, nrmat_ns, ipivArray,
                      Barray, nrmat_ns, &info, 1);

  // copy result into tau00
  // printf("copyTauToTau00\n");
  copyTauToTau00Cuda<<<blockSize,1>>>(devTau00, devTau, kkrzs_ns, nrmat_ns);
  // printf("transferMatrixFromGPU\n");
  transferMatrixFromGPUCuda(tau00, devTau00);
}

#ifndef ARCH_IBM
void solveTau00zzgesv_cusolver(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                               Complex *tMatrix, Complex *devM, Matrix<Complex> &tau00)
{
  hipsolverHandle_t cusolverDnHandle = DeviceStorage::getCusolverDnHandle();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK

  hipDoubleComplex *devTau = d.getDevTau();
  hipDoubleComplex *devTau00 = d.getDevTau00();
  
  zeroMatrixCuda(devTau, blockSize*numBlocks, kkrsz_ns);
  zeroMatrixCuda(deviceData.t, blockSize*numBlocks, kkrsz_ns);
  copyTMatrixToTauCuda<<<blockSize,1>>>(deviceData.t, tMatrix], kkrsz_ns, nrmat_ns);

  int info, iter;

  hipsolverStatus_t status = hipsolverDnZZgesv(cusolverDnHandle, nrmat_ns, kkrsz_ns,
                                             devM, nrmat_ns, deviceData.ipiv, deviceData.t, nrmat_ns, deviceData.tau, nrmat_ns,
    deviceData.work, deviceData.workBytes, &iter, deviceData.info);

  if(status!=HIPSOLVER_STATUS_SUCCESS)
  {
    printf("hipsolverDnZZgesv returned %d\n",status);
  }

  copyTauToTau00Cuda<<<blockSize,1>>>(deviceData.tau00, deviceData.tau, kkrsz_ns, nrmat_ns);
  transferMatrixFromGPU(tau00, deviceData.tau00);
}
#endif

void solveTau00zgetrf_cusolver(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                               Complex *tMatrix, Complex *devM, Matrix<Complex> &tau00)
{
  hipsolverHandle_t cusolverDnHandle = DeviceStorage::getCusolverDnHandle();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK

  zeroMatrixCuda(deviceData.tau, nrmat_ns, kkrsz_ns);
  copyTMatrixToTau<<<blockSize,1>>>(deviceData.tau, tMatrix, kkrsz_ns, nrmat_ns);

  hipsolverDnZgetrf(cusolverDnHandle, nrmat_ns, nrmat_ns, 
                   devM, nrmat_ns,
           (hipDoubleComplex *)deviceData.work,
           deviceData.ipiv,
           deviceData.info );

  hipsolverDnZgetrs(cusolverDnHandle, HIPBLAS_OP_N, nrmat_ns, kkrsz_ns,
      devM, nrmat_ns, deviceData.ipiv, deviceData.tau, nrmat_ns, deviceData.info);

  // copy result into tau00
  copyTauToTau00<<<blockSize,1>>>(deviceData.tau00, deviceData.tau, kkrsz_ns, nrmat_ns);
  transferMatrixFromGPU(tau00, deviceData.tau00);
}
